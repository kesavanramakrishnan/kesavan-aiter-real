// SPDX-License-Identifier: MIT
// Copyright (C) 2024-2025, Advanced Micro Devices, Inc. All rights reserved.
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include "gemm_moe_ck2stages_lookup.h"
#include "gemm_moe_ck2stages.h"
#include "gemm_moe_ck2stages_heuristic_dispatch.hpp"
#include "moe_ck.h"
#include <cmath>

using MoeKernelMap = std::unordered_map<std::string, MoeKernel>;

// API for user aiter.ck_moe_stage1(...)

template <int stage = 1>
MoeKernel moe_dispatch(std::string &kernelName, int block_m, int inter_dim)
{
    static const auto lookup = []
    {
        return MoeKernelMap{GENERATE_LOOKUP_TABLE()};
    }();

    if (kernelName != "")
    {
        auto it = lookup.find(kernelName);
        if (it != lookup.end())
        {
            auto kernel = it->second;
            return kernel;
        }
        std::cout << "[aiter] ck kernel not found: " << kernelName << std::endl;
    }
    if constexpr (stage == 1)
    {
        return moe_stage1_heuristic_dispatch(block_m);
    }
    else
    {
        return moe_stage2_heuristic_dispatch(block_m, inter_dim);
    }
}

void ck_moe_stage1(torch::Tensor &hidden_states,     // [m, k], input token
                   torch::Tensor &w1,                // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &w2,                // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &sorted_token_ids,  // [max_num_tokens_padded]
                   torch::Tensor &sorted_expert_ids, // [max_num_m_blocks]
                   torch::Tensor &num_valid_ids,     // [1]
                   torch::Tensor &out,               // [m * topk, inter_dim]
                   int topk,
                   std::string &kernelName,
                   std::optional<torch::Tensor> w1_scale = std::nullopt, // [e, 1, n], gate(up) scale
                   std::optional<torch::Tensor> a1_scale = std::nullopt, // [m, 1], token scale
                   std::optional<int> block_m = 32,
                   std::optional<torch::Tensor> sorted_weights = std::nullopt,
                   int quant_type = 0,
                   int activation = 0)
{
    const at::cuda::OptionalCUDAGuard device_guard(device_of(out));
    at::cuda::getCurrentCUDAStream().stream();

    TORCH_CHECK(out.dtype() == at::ScalarType::BFloat16 || out.dtype() == at::ScalarType::Half,
                "Out dtype only support BFloat16/Float16!")

    int tokens = hidden_states.size(0);
    int sorted_size = sorted_token_ids.size(0);
    int E = w1.size(0);
    int N = w1.size(1) / 2;
    int K = hidden_states.size(-1);
    int MPerBlock = block_m.value();

    void *hidden_states_ptr = hidden_states.data_ptr();
    void *w1_ptr = w1.transpose(1, 2).data_ptr();
    void *w2_ptr = w2.data_ptr();
    void *sorted_token_ids_ptr = sorted_token_ids.data_ptr();
    void *sorted_expert_ids_ptr = sorted_expert_ids.data_ptr();
    void *num_valid_ids_ptr = num_valid_ids.data_ptr();
    void *sorted_weights_ptr = sorted_weights.has_value() ? sorted_weights.value().data_ptr() : nullptr;
    void *out_ptr = out.data_ptr();
    void *w1_scale_ptr = w1_scale.has_value() ? w1_scale.value().data_ptr() : nullptr;
    void *a1_scale_ptr = a1_scale.has_value() ? a1_scale.value().data_ptr() : nullptr;
    if (!hidden_states_ptr || !w1_ptr || !w2_ptr || !sorted_token_ids_ptr || !sorted_expert_ids_ptr || !num_valid_ids_ptr || !out_ptr)
    {
        std::cerr << "detect null ptr !" << std::endl;
        return;
    }

    if (hidden_states.dtype() == at::ScalarType::Byte && w1.dtype() == at::ScalarType::Byte)
    {
        K *= 2;
    }

    auto kernel = moe_dispatch<1>(kernelName, MPerBlock, N);

    kernel(at::cuda::getCurrentCUDAStream().stream(),
           tokens, sorted_size, N, K, topk,
           hidden_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w1_scale_ptr, a1_scale_ptr);
}

void ck_moe_stage2(torch::Tensor &inter_states,      // [m, k], input token
                   torch::Tensor &w1,                // [e, n, k]/[e, 2*n, k], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &w2,                // [expert, dim, inter_dim], pre-shuffle([e, nr, kr, w])
                   torch::Tensor &sorted_token_ids,  // [max_num_tokens_padded]
                   torch::Tensor &sorted_expert_ids, // [max_num_m_blocks]
                   torch::Tensor &num_valid_ids,     // [1]
                   torch::Tensor &out,               // [max_num_tokens_padded, inter_dim]
                   int topk,
                   std::string &kernelName,
                   std::optional<torch::Tensor> w2_scale = std::nullopt, // [e, 1, n], gate(up) scale
                   std::optional<torch::Tensor> a2_scale = std::nullopt, // [m, 1], token scale
                   std::optional<int> block_m = 32,
                   std::optional<torch::Tensor> sorted_weights = std::nullopt,
                   int quant_type = 0,
                   int activation = 0)
{
    TORCH_CHECK(out.dtype() == at::ScalarType::BFloat16 || out.dtype() == at::ScalarType::Half,
                "Out dtype only support BFloat16/Float16!")

    int tokens = inter_states.size(0);
    int sorted_size = sorted_token_ids.size(0);
    int E = w1.size(0);
    int N = w2.size(1);
    int K = inter_states.size(-1);
    int MPerBlock = block_m.value();

    void *inter_states_ptr = inter_states.data_ptr();
    void *w1_ptr = w1.data_ptr();
    void *w2_ptr = w2.data_ptr();
    void *sorted_token_ids_ptr = sorted_token_ids.data_ptr();
    void *sorted_expert_ids_ptr = sorted_expert_ids.data_ptr();
    void *sorted_weights_ptr = sorted_weights.has_value() ? sorted_weights.value().data_ptr() : nullptr;
    void *num_valid_ids_ptr = num_valid_ids.data_ptr();
    void *out_ptr = out.data_ptr();
    void *w2_scale_ptr = w2_scale.has_value() ? w2_scale.value().data_ptr() : nullptr;
    void *a2_scale_ptr = a2_scale.has_value() ? a2_scale.value().data_ptr() : nullptr;
    if (!inter_states_ptr || !w1_ptr || !w2_ptr || !sorted_token_ids_ptr || !sorted_expert_ids_ptr || !num_valid_ids_ptr || !out_ptr)
    {
        std::cerr << "detect null ptr !" << std::endl;
        return;
    }
    if (inter_states.dtype() == at::ScalarType::Byte && w2.dtype() == at::ScalarType::Byte)
    {
        K *= 2;
    }
    auto kernel = moe_dispatch<2>(kernelName, MPerBlock, K);

    kernel(at::cuda::getCurrentCUDAStream().stream(),
           tokens, sorted_size, N, K, topk,
           inter_states_ptr, w1_ptr, w2_ptr, sorted_token_ids_ptr, sorted_expert_ids_ptr, sorted_weights_ptr, num_valid_ids_ptr, out_ptr, w2_scale_ptr, a2_scale_ptr);
}
#include "hip/hip_runtime.h"
// SPDX-License-Identifier: MIT
// Copyright (C) 2024-2025, Advanced Micro Devices, Inc. All rights reserved.
/*
 * @Script: topk_softmax_kernels_group.cu
 * @Author: valarLip
 * @Email: lingpeng.jin@amd.com
 * @Create At: 2025-03-01 12:16:14
 * @Last Modified By: valarLip
 * @Last Modified At: 2025-05-02 15:52:13
 * @Description: This is description.
 */

#include <hip/hip_runtime.h>
#include <torch/all.h>
#include <ATen/hip/HIPContext.h>
#include <ATen/hip/impl/HIPGuardImplMasqueradingAsCUDA.h>
#include "dispatch_utils.h"
#include "py_itfs_common.h"
#include <hipcub/util_type.hpp>
#include <hipcub/hipcub.hpp>

#define WARP_SIZE 64
namespace aiter
{
    template <typename T, typename F>
    __device__ constexpr T wave_reduce(T local, F reduce_f)
    {
        constexpr int reduce_stage = 6; // 1<<6=64
        T v_local = local;
#pragma unroll
        for (int i_stage = 0; i_stage < reduce_stage; i_stage++)
        {
            int src_lane = __lane_id() ^ (1 << i_stage);
            int32_t v_remote_tmp =
                __builtin_amdgcn_ds_bpermute(src_lane << 2, __builtin_bit_cast(int32_t, v_local));
            T v_remote = __builtin_bit_cast(T, v_remote_tmp);
            v_local = reduce_f(v_local, v_remote);
        }
        return v_local;
    }

    __inline__ __device__ void warpReduceMax(float &val, int &idx)
    {
        static_assert(64 == WARP_SIZE, "WARP_SIZE == 64");
#pragma unroll
        for (int i = 0; i < 6; i++)
        {
            int offset = 1 << i;
            float tmp_val = __shfl_down(val, offset);
            int tmp_idx = __shfl_down(idx, offset);
            if (tmp_val > val)
            {
                val = tmp_val;
                idx = tmp_idx;
            }
        }
    }

    __device__ void blockReduceMax(float &val, int &idx)
    {
        __shared__ float shared_vals[32];
        __shared__ int shared_idxs[32];

        int lane = threadIdx.x % WARP_SIZE;
        int wid = threadIdx.x / WARP_SIZE;

        warpReduceMax(val, idx);

        if (lane == 0)
        {
            shared_vals[wid] = val;
            shared_idxs[wid] = idx;
        }
        __syncthreads();

        if (wid == 0)
        {
            val = (lane < (blockDim.x + WARP_SIZE - 1) / WARP_SIZE) ? shared_vals[lane] : -INFINITY;
            idx = (lane < (blockDim.x + WARP_SIZE - 1) / WARP_SIZE) ? shared_idxs[lane] : -1;

            warpReduceMax(val, idx);
        }
        __syncthreads();
    }

    template <typename DTYPE_I, typename f32vec, int NUM_GRP, bool need_renorm, bool isBiased, bool isSoftmax>
    __global__ void grouped_topk_kernel(
        DTYPE_I *__restrict__ gating_output,         // [num_tokens, hidden_size]
        const DTYPE_I *__restrict__ correction_bias, // [num_expert]
        float *__restrict__ topk_weights,            // [num_tokens, topk]
        int *__restrict__ topk_ids,                  // [num_tokens, topk]
        const size_t stride_tk,
        const int num_experts,
        const int topk,
        const int topk_group,
        const int num_tokens,
        const float routed_scaling_factor)
    {
        static_assert(NUM_GRP <= WARP_SIZE, "NUM_GRP must be <= WARP_SIZE");
        // 256 E, 8->4 group, 32 e/group
        const int experts_per_group = num_experts / NUM_GRP;
        extern __shared__ char shared_mem[];
        const int token_idx = blockIdx.x;

        char *ptr = (char *)(((size_t)shared_mem + 255) & ~255);
        float *scores = reinterpret_cast<float *>(ptr);
        ptr += num_experts * sizeof(float);

        float *group_scores = reinterpret_cast<float *>(ptr);
        ptr += NUM_GRP * sizeof(float);

        int *topk_indices = reinterpret_cast<int *>(ptr);
        ptr += topk * sizeof(int);

        float *topk_values = reinterpret_cast<float *>(ptr);
        // ptr += topk * sizeof(float);

        // int *topk_indices_f = reinterpret_cast<int *>(ptr);
        // ptr += topk * sizeof(int);

        // float *topk_values_f = reinterpret_cast<float *>(ptr);

        f32vec *scores_vec = reinterpret_cast<f32vec *>(scores);
        using cktype_i = typename t2ck<DTYPE_I>::type;
        static constexpr int vec_size = ck_tile::vector_traits<f32vec>::vector_size;
        using vec_i = ck_tile::ext_vector_t<cktype_i, vec_size>;
        const int num_experts_vec = num_experts / vec_size;

        if constexpr (!isSoftmax)
        {
            auto const *input_ptr = gating_output + token_idx * num_experts;
            for (int e = threadIdx.x; e < num_experts_vec; e += blockDim.x)
            {
                vec_i tmp = reinterpret_cast<vec_i const *>(input_ptr)[e];
                vec_i tmp2;
                if constexpr (isBiased)
                    tmp2 = reinterpret_cast<vec_i const *>(correction_bias)[e];
                f32vec gating;
#pragma unroll
                for (size_t i = 0; i < vec_size; i++)
                {
                    gating[i] = ck_tile::type_convert<float>(tmp[i]);
                    gating[i] = 1.0f / (1.0f + expf(-gating[i]));
                    if constexpr (isBiased)
                    {
                        gating[i] += ck_tile::type_convert<float>(tmp2[i]);
                    }
                }
                scores_vec[e] = gating;
            }
            __syncthreads();
        }
        else
        {
            __shared__ float sdata;
            float max_val = -INFINITY;
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {

                float gating = gating_output[token_idx * num_experts + e];
                scores[e] = gating;
                if (gating > max_val)
                {
                    max_val = gating;
                }
            }
            __syncthreads();
#pragma unroll
            for (int i = 0; i < 6; i++)
            {
                int offset = 1 << i;
                float tmp_val = __shfl_down(max_val, offset);
                if (tmp_val > max_val)
                {
                    max_val = tmp_val;
                }
            }
            if (threadIdx.x == 0)
            {
                sdata = max_val;
            }
            __syncthreads();
            max_val = sdata;
            float thread_sum = 0.0;
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {
                scores[e] = expf(scores[e] - max_val);
                thread_sum += scores[e];
            }
            __syncthreads();
            thread_sum = wave_reduce(thread_sum, [](float a, float b)
                                     { return a + b; });
            for (int e = threadIdx.x; e < num_experts; e += blockDim.x)
            {
                scores[e] /= thread_sum;
            }
            __syncthreads();
        }

        if constexpr (isBiased)
        {
            for (int g = threadIdx.x; g < NUM_GRP; g += blockDim.x)
            {
                float max1 = -INFINITY, max2 = -INFINITY;
                const int start = g * experts_per_group;
                const int end = start + experts_per_group;

                for (int e = start; e < end; ++e)
                {
                    if (scores[e] > max1)
                    {
                        max2 = max1;
                        max1 = scores[e];
                    }
                    else if (scores[e] > max2)
                    {
                        max2 = scores[e];
                    }
                }
                group_scores[g] = max1 + max2;
            }
            __syncthreads();
        }
        else
        {
#pragma unroll
            for (int g = threadIdx.x; g < NUM_GRP; g += blockDim.x)
            {
                float max1 = -INFINITY;
                const int start = g * experts_per_group;
                const int end = start + experts_per_group;
                for (int e = start; e < end; ++e)
                {
                    if (scores[e] > max1)
                    {
                        max1 = scores[e];
                    }
                }
                group_scores[g] = max1;
            }
            __syncthreads();
        }

        for (int k = 0; k < topk_group; k++)
        {
            float max_val = -INFINITY;
            int max_idx = NUM_GRP;
#pragma unroll
            for (int g = 0; g < NUM_GRP; g++)
            {
                if (group_scores[g] > max_val)
                {
                    max_val = group_scores[g];
                    max_idx = g;
                }
            }
            group_scores[max_idx] = -INFINITY;
        }

        for (int e = threadIdx.x; e < num_experts_vec; e += blockDim.x)
        {
            int group_idx = e * vec_size / experts_per_group;
            if (group_scores[group_idx] != -INFINITY)
            {
                scores_vec[e] = -INFINITY;
            }
        }
        __syncthreads();

        using kvp = hipcub::KeyValuePair<int, float>;
        using BlockReduce = hipcub::BlockReduce<kvp, WARP_SIZE>;
        __shared__ typename BlockReduce::TempStorage tmpStorage;
        kvp thread_kvp;
        hipcub::ArgMax arg_max;

        float sum = 0.0f;
        for (int k = 0; k < topk; ++k)
        {
            float max_val = scores[k];
            int max_idx = k;

            for (int e = threadIdx.x; e < num_experts_vec; e += blockDim.x)
            {
                f32vec tmp = scores_vec[e];
#pragma unroll
                for (size_t i = 0; i < vec_size; i++)
                {
                    if (tmp[i] > max_val)
                    {
                        max_val = tmp[i];
                        max_idx = e * vec_size + i;
                    }
                }
            }
            thread_kvp.key = max_idx;
            thread_kvp.value = max_val;
            const kvp result_kvp = BlockReduce(tmpStorage).Reduce(thread_kvp, arg_max);
            // warpReduceMax(max_val, max_idx);
            // blockReduceMax(max_val, max_idx);

            if (threadIdx.x == 0)
            {
                max_val = result_kvp.value;
                max_idx = result_kvp.key;
                if constexpr (isBiased)
                {
                    max_val -= correction_bias[max_idx];
                }
                scores[max_idx] = -INFINITY;
                topk_indices[k] = max_idx;
                topk_values[k] = max_val;
                if (need_renorm)
                {
                    sum += max_val;
                }
            }
            __syncthreads();
        }

        if (need_renorm)
        {
            if (threadIdx.x == 0)
            {
                scores[0] = routed_scaling_factor / sum; // reuse lds
            }
            __syncthreads();
            sum = scores[0];
        }
        else
        {
            sum = routed_scaling_factor;
        }

        for (int k = threadIdx.x; k < topk; k += blockDim.x)
        {
            topk_weights[token_idx * stride_tk + k] = topk_values[k] * sum;
            topk_ids[token_idx * stride_tk + k] = topk_indices[k];
        }
    }
} // namespace aiter end

#define LAUNCH_KERNEL()                                    \
    switch (num_experts % 4)                               \
    {                                                      \
    case 0:                                                \
        using vec4_type = ck_tile::ext_vector_t<float, 4>; \
        LAUNCHER2(vec4_type)                               \
        break;                                             \
    case 2:                                                \
        using vec2_type = ck_tile::ext_vector_t<float, 2>; \
        LAUNCHER2(vec2_type)                               \
        break;                                             \
    default:                                               \
        using vec1_type = ck_tile::ext_vector_t<float, 1>; \
        LAUNCHER2(vec1_type)                               \
        break;                                             \
    }
#define LAUNCHER2(VEC_F)                                                        \
    switch (num_expert_group)                                                   \
    {                                                                           \
    case 8:                                                                     \
        LAUNCHER3(VEC_F, 8)                                                     \
        break;                                                                  \
    case 4:                                                                     \
        LAUNCHER3(VEC_F, 4)                                                     \
        break;                                                                  \
    case 2:                                                                     \
        LAUNCHER3(VEC_F, 2)                                                     \
        break;                                                                  \
    case 1:                                                                     \
        LAUNCHER3(VEC_F, 1)                                                     \
        break;                                                                  \
    default:                                                                    \
        TORCH_CHECK(false, "Unsupported num_expert_group: ", num_expert_group); \
        break;                                                                  \
    }
#define LAUNCHER3(VEC_F, NUM_GRP)        \
    switch (need_renorm)                 \
    {                                    \
    case true:                           \
        LAUNCHER4(VEC_F, NUM_GRP, true)  \
        break;                           \
    default:                             \
        LAUNCHER4(VEC_F, NUM_GRP, false) \
    }

#define LAUNCHER4(VEC_F, NUM_GRP, need_renorm)                                        \
    if constexpr (isBiased)                                                           \
    {                                                                                 \
        LAUNCHER_biased_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, true, false) \
    }                                                                                 \
    else                                                                              \
    {                                                                                 \
        if (isSoftmax)                                                                \
        {                                                                             \
            LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, false, true)    \
        }                                                                             \
        else                                                                          \
        {                                                                             \
            LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, false, false)   \
        }                                                                             \
    }

#define LAUNCHER_biased_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax)                                                                            \
    VLLM_DISPATCH_FLOATING_TYPES(                                                                                                                                        \
        gating_output.scalar_type(), "biased_grouped_topk_kernel", [&]                                                                                                   \
        { hipLaunchKernelGGL((aiter::grouped_topk_kernel<scalar_t, VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax>), dim3(grid), dim3(block), shared_mem_size, stream, \
                             gating_output.data_ptr<scalar_t>(),                                                                                                         \
                             correction_bias.data_ptr<scalar_t>(),                                                                                                       \
                             topk_weights.data_ptr<float>(),                                                                                                             \
                             topk_ids.data_ptr<int>(),                                                                                                                   \
                             stride_tk,                                                                                                                                  \
                             num_experts,                                                                                                                                \
                             topk,                                                                                                                                       \
                             topk_grp, num_tokens, routed_scaling_factor); });

#define LAUNCHER_grouped_topk_kernel(VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax)                                                                                   \
    VLLM_DISPATCH_FLOATING_TYPES(                                                                                                                                        \
        gating_output.scalar_type(), "grouped_topk_kernel", [&]                                                                                                          \
        { hipLaunchKernelGGL((aiter::grouped_topk_kernel<scalar_t, VEC_F, NUM_GRP, need_renorm, isBiased, isSoftmax>), dim3(grid), dim3(block), shared_mem_size, stream, \
                             gating_output.data_ptr<scalar_t>(),                                                                                                         \
                             nullptr,                                                                                                                                    \
                             topk_weights.data_ptr<float>(),                                                                                                             \
                             topk_ids.data_ptr<int>(),                                                                                                                   \
                             stride_tk,                                                                                                                                  \
                             num_experts,                                                                                                                                \
                             topk,                                                                                                                                       \
                             topk_grp, num_tokens, routed_scaling_factor); });

void biased_grouped_topk(
    torch::Tensor &gating_output,   // [num_tokens, num_experts]
    torch::Tensor &correction_bias, // [num_expert]
    torch::Tensor &topk_weights,    // [num_tokens, topk]
    torch::Tensor &topk_ids,        // [num_tokens, topk]
    int num_expert_group,
    int topk_grp,
    bool need_renorm,
    const float routed_scaling_factor = 1.)
{
    const bool isBiased = true;
    bool isSoftmax = false;
    int num_tokens = gating_output.size(0);
    int num_experts = gating_output.size(1);
    int topk = topk_ids.size(1);
    size_t stride_tk = topk_ids.stride(0);
    TORCH_CHECK(stride_tk == topk_weights.stride(0), "topk_ids.stride(0) == topk_weights.stride(0)");
    TORCH_CHECK(gating_output.dtype() == correction_bias.dtype(), "gating_output.dtype() == correction_bias.dtype()");

    dim3 grid(num_tokens);
    dim3 block(64);
    size_t shared_mem_size = (num_experts * sizeof(float) +
                              (num_expert_group + 1) * sizeof(float) +
                              topk * sizeof(int) +
                              topk * sizeof(float) + 255) &
                             ~255;

    const at::hip::OptionalHIPGuardMasqueradingAsCUDA device_guard(device_of(gating_output));
    const hipStream_t stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    LAUNCH_KERNEL()
}

void grouped_topk(
    torch::Tensor &gating_output, // [num_tokens, num_experts]
    torch::Tensor &topk_weights,  // [num_tokens, topk]
    torch::Tensor &topk_ids,      // [num_tokens, topk]
    int num_expert_group,
    int topk_grp,
    bool need_renorm,
    bool scoring_func = true,
    const float routed_scaling_factor = 1.)
{
    const bool isBiased = false;
    bool isSoftmax = scoring_func;
    int num_tokens = gating_output.size(0);
    int num_experts = gating_output.size(1);
    int topk = topk_ids.size(1);
    size_t stride_tk = topk_ids.stride(0);
    auto correction_bias = topk_ids;
    TORCH_CHECK(stride_tk == topk_weights.stride(0), "topk_ids.stride(0) == topk_weights.stride(0)");

    dim3 grid(num_tokens);
    dim3 block(64);
    size_t shared_mem_size = (num_experts * sizeof(float) +
                              (num_expert_group + 1) * sizeof(float) +
                              topk * sizeof(int) +
                              topk * sizeof(float) + 255) &
                             ~255;

    const at::hip::OptionalHIPGuardMasqueradingAsCUDA device_guard(device_of(gating_output));
    const hipStream_t stream = at::hip::getCurrentHIPStreamMasqueradingAsCUDA();

    LAUNCH_KERNEL()
}

#undef LAUNCHER4
#undef LAUNCHER3
#undef LAUNCHER2
#undef LAUNCH_KERNEL
